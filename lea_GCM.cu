#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <crtdbg.h>
#include <corecrt_malloc.h>


#pragma once
typedef unsigned char byte;
typedef unsigned int uint32;

#define GETU32(p)(((uint32)p[0]<<24)^((uint32)p[1]<<16)^((uint32)p[2]<<8)^((uint32)p[3]))
#define PUTU32(b, x) { \
(b)[0] = (byte)((x) >> 24); \
(b)[1] = (byte)((x) >> 16); \
(b)[2] = (byte)((x) >> 8); \
(b)[3] = (byte)(x); }

uint32 keyNum[8] = { 0xc3efe9db, 0x44626b02, 0x79e27c8a, 0x78df30ec, 0x715ea49e, 0xc785da0a, 0xe04ef22a, 0xe5c40957 };



// ����Ʈ �迭 �����ϱ� (�޸� Ȯ���� �Լ� ȣ�� ���� �̸�)
// src[] --> desr[] (�迭�� ũ��: len) 
void copy_b_array(byte src[], int len, byte dest[]) {
    for (int i = 0; i < len; i++) {
        dest[i] = src[i];
    }
}

// ����Ʈ �迭�� �ٸ� �迭�� ���� XOR �ϱ�
// ����: X <-- X xor Y (�迭 X�� ������Ʈ �ϴ� ���)
// data[] <-- data[] xor xor_arr[]
void xor_b_array(byte data[], int len, byte xor_arr[]) {
    for (int i = 0; i < len; i++) {
        data[i] ^= xor_arr[i]; // data[]�� ������Ʈ
    }
}

void byte2state(uint32* state, byte* b) {

    state[0] = *((uint32*)b + 0);
    state[1] = *(uint32*)(b + 4);
    state[2] = *(uint32*)(b + 8);
    state[3] = *(uint32*)(b + 12);

}
void state2byte(uint32 state[4], byte b[16]) {
    /*
    PUTU32(b, state[0]);
    PUTU32(b + 4, state[1]);
    PUTU32(b + 8, state[2]);
    PUTU32(b + 12, state[3]);
    */
    *(uint32*)(b + 0) = state[0];
    *(uint32*)(b + 4) = state[1];
    *(uint32*)(b + 8) = state[2];
    *(uint32*)(b + 12) = state[3];
}
uint32 rotLeft(uint32 u, int k) {
    u = (u << k) | (u >> (32 - k));
    return u;
}
uint32 rotRight(uint32 u, int k) {
    u = (u >> k) | (u << (32 - k));
    return u;
}

void LEA32_Encrypt(byte* pt, uint32 rk[24][6], byte* ct)
{
    uint32 state[4] = { 0x00, };
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;
    //std::cout <<"��ȣȭ ����" << std::endl;
    uint32 temp = 0x00;
    /*
    std::cout << "�ʱ� �� ���" << std::endl;
    for (int i = 0; i < 16; i++) {
       printf("%02x, ", pt[i]);
    }*/
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;

    byte2state(state, pt);
    for (int i = 0; i < 24; i++) {
        temp = state[0];
        state[0] = rotLeft((state[0] ^ rk[i][0]) + ((state[1] ^ rk[i][1])), 9);
        state[1] = rotRight((state[1] ^ rk[i][2]) + ((state[2] ^ rk[i][3])), 5);
        state[2] = rotRight((state[2] ^ rk[i][4]) + ((state[3] ^ rk[i][5])), 3);
        state[3] = temp;
        //printf("��ȣȭ %d��° ���� ��� :   %08x,   %08x,   %08x,   %08x", i, state[0], state[1], state[2], state[3]);
        //std::cout << std::endl;
    }
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;

    state2byte(state, ct);
    state2byte(state, pt);
    /*
    std::cout << "������ȣ��" << std::endl;
    for (int i = 0; i < 16; i++) {
       printf("%02x,  ",ct[i]);
    }
    std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;
    */
}

void leaDec(byte* pt, uint32 rk[24][6], byte* ct)
{
    //keyExpension(firkey,src);
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;
    //std::cout << "��ȣȭ ����" << std::endl;
    uint32 temp[4] = { 0x00, };
    uint32 state[4] = { 0x00 };
    byte2state(state, ct);

    for (int i = 0; i < 24; i++) {//24-i�� �� ����
        temp[0] = state[0];
        temp[1] = state[1];
        temp[2] = state[2];
        temp[3] = state[3];
        state[0] = temp[3];//ct�� 3��° 32 ����� state[0]�� ��
        state[1] = (rotRight(temp[0], 9) - (state[0] ^ rk[(23 - i)][0])) ^ rk[(23 - i)][1];
        state[2] = (rotLeft(temp[1], 5) - (state[1] ^ rk[(23 - i)][2]) ^ rk[(23 - i)][3]);
        state[3] = (rotLeft(temp[2], 3) - (state[2] ^ rk[(23 - i)][4]) ^ rk[(23 - i)][5]);

        //printf("��ȣȭ %d��° ���� ��� :   %08x,   %08x,   %08x,   %08x", i, state[0], state[1], state[2], state[3]);
        //std::cout << std::endl;
    }
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;

    state2byte(state, pt);
    state2byte(state, ct);

    /*
    std::cout << "������ȭȭ�� ��" << std::endl;
    for (int i = 0; i < 16; i++) {
       printf("%02x,  ", pt[i]);
    }*/
    //std::cout << std::endl; std::cout << std::endl; std::cout << std::endl;

}

void LEA32_Enc_KeySchedule(byte firkey[16], uint32 rk[24][6])//24*4���� Ű �� ���忡 4��
{
    uint32 k[96] = { 0x00 };
    uint32 sk[4] = { 0x00 };
    //make byte plaintext to uint32
    byte2state(sk, firkey);
    //printf("\nfirkey = %08X, %08X, %08X, %08X\n", sk[0], sk[1], sk[2], sk[3]);
    /*
    for (int i = 0; i < 8; i++) {
       printf("****%02x", keyNum[i]);
    }*/
    /*
    for (int i = 0; i < 4; i++) {
       printf("****%02x", sk[i]);
    }*/
    //std::cout << "Ű ��ü ��� " << std::endl;
    for (int i = 0; i < 24; i++) {
        sk[0] = rotLeft((uint32)(sk[0] + rotLeft(keyNum[i % 4], i)), 1);
        sk[1] = rotLeft((uint32)(sk[1] + rotLeft(keyNum[i % 4], i + 1)), 3);
        sk[2] = rotLeft((uint32)(sk[2] + rotLeft(keyNum[i % 4], i + 2)), 6);
        sk[3] = rotLeft((uint32)(sk[3] + rotLeft(keyNum[i % 4], i + 3)), 11);
        rk[i][0] = sk[0];
        rk[i][1] = sk[1];
        rk[i][2] = sk[2];
        rk[i][3] = sk[1];
        rk[i][4] = sk[3];
        rk[i][5] = sk[1];
        //printf("%d��° Ű :   %08x,   %08x,   %08x,   %08x,   %08x,   %08x ",i, roundKey[6*i], roundKey[6*i+1], roundKey[6*i+2], roundKey[6*i+3], roundKey[6*i + 4], roundKey[6*i + 5]);
        //std::cout << std::endl;
    }
}

byte R0[256] = {
0x00, 0x01, 0x03, 0x02, 0x07, 0x06, 0x04, 0x05, 0x0e, 0x0f, 0x0d, 0x0c, 0x09, 0x08, 0x0a, 0x0b,
0x1c, 0x1d, 0x1f, 0x1e, 0x1b, 0x1a, 0x18, 0x19, 0x12, 0x13, 0x11, 0x10, 0x15, 0x14, 0x16, 0x17,
0x38, 0x39, 0x3b, 0x3a, 0x3f, 0x3e, 0x3c, 0x3d, 0x36, 0x37, 0x35, 0x34, 0x31, 0x30, 0x32, 0x33,
0x24, 0x25, 0x27, 0x26, 0x23, 0x22, 0x20, 0x21, 0x2a, 0x2b, 0x29, 0x28, 0x2d, 0x2c, 0x2e, 0x2f,
0x70, 0x71, 0x73, 0x72, 0x77, 0x76, 0x74, 0x75, 0x7e, 0x7f, 0x7d, 0x7c, 0x79, 0x78, 0x7a, 0x7b,
0x6c, 0x6d, 0x6f, 0x6e, 0x6b, 0x6a, 0x68, 0x69, 0x62, 0x63, 0x61, 0x60, 0x65, 0x64, 0x66, 0x67,
0x48, 0x49, 0x4b, 0x4a, 0x4f, 0x4e, 0x4c, 0x4d, 0x46, 0x47, 0x45, 0x44, 0x41, 0x40, 0x42, 0x43,
0x54, 0x55, 0x57, 0x56, 0x53, 0x52, 0x50, 0x51, 0x5a, 0x5b, 0x59, 0x58, 0x5d, 0x5c, 0x5e, 0x5f,
0xe1, 0xe0, 0xe2, 0xe3, 0xe6, 0xe7, 0xe5, 0xe4, 0xef, 0xee, 0xec, 0xed, 0xe8, 0xe9, 0xeb, 0xea,
0xfd, 0xfc, 0xfe, 0xff, 0xfa, 0xfb, 0xf9, 0xf8, 0xf3, 0xf2, 0xf0, 0xf1, 0xf4, 0xf5, 0xf7, 0xf6,
0xd9, 0xd8, 0xda, 0xdb, 0xde, 0xdf, 0xdd, 0xdc, 0xd7, 0xd6, 0xd4, 0xd5, 0xd0, 0xd1, 0xd3, 0xd2,
0xc5, 0xc4, 0xc6, 0xc7, 0xc2, 0xc3, 0xc1, 0xc0, 0xcb, 0xca, 0xc8, 0xc9, 0xcc, 0xcd, 0xcf, 0xce,
0x91, 0x90, 0x92, 0x93, 0x96, 0x97, 0x95, 0x94, 0x9f, 0x9e, 0x9c, 0x9d, 0x98, 0x99, 0x9b, 0x9a,
0x8d, 0x8c, 0x8e, 0x8f, 0x8a, 0x8b, 0x89, 0x88, 0x83, 0x82, 0x80, 0x81, 0x84, 0x85, 0x87, 0x86,
0xa9, 0xa8, 0xaa, 0xab, 0xae, 0xaf, 0xad, 0xac, 0xa7, 0xa6, 0xa4, 0xa5, 0xa0, 0xa1, 0xa3, 0xa2,
0xb5, 0xb4, 0xb6, 0xb7, 0xb2, 0xb3, 0xb1, 0xb0, 0xbb, 0xba, 0xb8, 0xb9, 0xbc, 0xbd, 0xbf, 0xbe
};

byte R1[256] = {
0x00, 0xc2, 0x84, 0x46, 0x08, 0xca, 0x8c, 0x4e, 0x10, 0xd2, 0x94, 0x56, 0x18, 0xda, 0x9c, 0x5e,
0x20, 0xe2, 0xa4, 0x66, 0x28, 0xea, 0xac, 0x6e, 0x30, 0xf2, 0xb4, 0x76, 0x38, 0xfa, 0xbc, 0x7e,
0x40, 0x82, 0xc4, 0x06, 0x48, 0x8a, 0xcc, 0x0e, 0x50, 0x92, 0xd4, 0x16, 0x58, 0x9a, 0xdc, 0x1e,
0x60, 0xa2, 0xe4, 0x26, 0x68, 0xaa, 0xec, 0x2e, 0x70, 0xb2, 0xf4, 0x36, 0x78, 0xba, 0xfc, 0x3e,
0x80, 0x42, 0x04, 0xc6, 0x88, 0x4a, 0x0c, 0xce, 0x90, 0x52, 0x14, 0xd6, 0x98, 0x5a, 0x1c, 0xde,
0xa0, 0x62, 0x24, 0xe6, 0xa8, 0x6a, 0x2c, 0xee, 0xb0, 0x72, 0x34, 0xf6, 0xb8, 0x7a, 0x3c, 0xfe,
0xc0, 0x02, 0x44, 0x86, 0xc8, 0x0a, 0x4c, 0x8e, 0xd0, 0x12, 0x54, 0x96, 0xd8, 0x1a, 0x5c, 0x9e,
0xe0, 0x22, 0x64, 0xa6, 0xe8, 0x2a, 0x6c, 0xae, 0xf0, 0x32, 0x74, 0xb6, 0xf8, 0x3a, 0x7c, 0xbe,
0x00, 0xc2, 0x84, 0x46, 0x08, 0xca, 0x8c, 0x4e, 0x10, 0xd2, 0x94, 0x56, 0x18, 0xda, 0x9c, 0x5e,
0x20, 0xe2, 0xa4, 0x66, 0x28, 0xea, 0xac, 0x6e, 0x30, 0xf2, 0xb4, 0x76, 0x38, 0xfa, 0xbc, 0x7e,
0x40, 0x82, 0xc4, 0x06, 0x48, 0x8a, 0xcc, 0x0e, 0x50, 0x92, 0xd4, 0x16, 0x58, 0x9a, 0xdc, 0x1e,
0x60, 0xa2, 0xe4, 0x26, 0x68, 0xaa, 0xec, 0x2e, 0x70, 0xb2, 0xf4, 0x36, 0x78, 0xba, 0xfc, 0x3e,
0x80, 0x42, 0x04, 0xc6, 0x88, 0x4a, 0x0c, 0xce, 0x90, 0x52, 0x14, 0xd6, 0x98, 0x5a, 0x1c, 0xde,
0xa0, 0x62, 0x24, 0xe6, 0xa8, 0x6a, 0x2c, 0xee, 0xb0, 0x72, 0x34, 0xf6, 0xb8, 0x7a, 0x3c, 0xfe,
0xc0, 0x02, 0x44, 0x86, 0xc8, 0x0a, 0x4c, 0x8e, 0xd0, 0x12, 0x54, 0x96, 0xd8, 0x1a, 0x5c, 0x9e,
0xe0, 0x22, 0x64, 0xa6, 0xe8, 0x2a, 0x6c, 0xae, 0xf0, 0x32, 0x74, 0xb6, 0xf8, 0x3a, 0x7c, 0xbe
};

// GCM ǥ�ع����� Inc_32() �Լ�
// counter: (msb)  c[0] c[1] ... c[15] (lsb)
void counter_inc(byte counter[16]) {
    for (int i = 15; i >= 0; i--) { // c[15] --> c[0]
        if (counter[i] != 0xff) { //�ڸ��ø� ����
            counter[i]++;
            break; // for-loop�� ���
        }
        else { // 0xff --> 0x00, �ڸ��ø�
            counter[i] = 0x00;
        }
    }
}

//AES CTR mode
void AES_CTR(byte PT[], int pt_len, byte key[16], byte CTR[16], byte CT[]) {
    int num_blocks, remainder;
    num_blocks = pt_len / 16;
    remainder = pt_len - num_blocks * 16;

    byte pt[16], ctr_ct[16];
    uint32 rk[24][6];
    byte current_ctr[16];

    LEA32_Enc_KeySchedule(key, rk);

    copy_b_array(CTR, 16, current_ctr);
    for (int i = 0; i < num_blocks; i++) {
        for (int j = 0; j < 16; j++) pt[j] = PT[i * 16 + j];
        LEA32_Encrypt(current_ctr, rk, ctr_ct);
        xor_b_array(pt, 16, ctr_ct); //pt�� ��ȣ��
        for (int j = 0; j < 16; j++) CT[i * 16 + j] = pt[j];
        counter_inc(current_ctr);
    }
    LEA32_Encrypt(current_ctr, rk, ctr_ct);
    for (int i = 0; i < remainder; i++) {
        pt[i] = PT[16 * num_blocks + i];
        pt[i] ^= ctr_ct[i];
        CT[16 * num_blocks + i] = pt[i];
    }
}

// GF(2^128)�� xtime(), m(x) = 1 + x + x^2 + x^7 + x^128
// p(x) * x  = (p0 + p1*x + p2*x^2 + ... + p127*x^127)*x
//           = p0*x + p1*x^2 + ... + p127*x^128
//           = p0*x + p1*x^2 + ... p126*x^127 + p127*(1+x+x^2+x^7)
//           = [0, p0, p1, ... , p126] xor p127*[1110 0001 000....]
// ����: p(x) <-- x*p(x) �� p(x)�� ������Ʈ�ϴ� �����
//     [p0...p7] [p8...p15] ... [p120...p127]
// ==> [0p0..p6] [p7...p14] ... [p119...p126]
void GF128_xtime(byte p[16]) {
    //[1] �Լ� ���� ä���
    byte msb; //msb = p127
    msb = (byte)(p[15] & 0x01);
    for (int i = 15; i > 0; i--) {
        // ... [...a] [bcde fghi] ==> ... [ ] [abcd efgh] ...
        p[i] = (p[i] >> 1) | ((p[i - 1] & 0x01) << 7);
    }
    p[0] >>= 1; //p[0] = p[0] >> 1;
    if (msb != 0) { // p127=1
        p[0] ^= 0xe1; // p[0] = p[0] ^ 0b11100001;
    }
}

//== (GHASH) =====
// HTable�� �̿��� GF(2^128) ����: p(x) <-- p(x)*q(x)
// p(x), q(x) = q0 + q1*x + q2*x^2 + ... + q127*x^127
// p(x)*q(x) = p(x)*(q0 + q1*x + q2*x^2 + ... + q127*x^127)
//     = q0*p(x) + q1*x*p(x) + q2*x^2*p(x) + ... + q127*x^127*p(x)
/*
   H(x)*p(x) = H(x)*(P[0] + P[1]*x^8 + P[2]*x^16 + ... + P[15]*x^120)
    = H(x)*P[0] + (H(x)*P[1] + ... (H(x)*P[13] + (H(x)*P[14] + (H(x)*P[15])*x^8)*x^8)*x^8) ... )*x^8)
*/
void GF128_Hmul(byte state[16], byte HT[256][16], byte R0[256], byte R1[256]) {
    byte W[16] = { 0, };
    byte temp;
    byte in[16];

    //for (int j = 0; j < 16; j++) {
    //   in[j] = state[j];
    //}
    copy_b_array(state, 16, in);
    for (int i = 0; i < 15; i++) { // 0, 1, ... , 14
        temp = in[15 - i];  // temp: in[15], in[14], ... in[1] (7�� ���� ���׽�)
        for (int j = 0; j < 16; j++) {
            W[j] ^= HT[temp][j];
        }
        //xor_b_array(W, 16, HT[temp]); // W ^= H(x)*in[15-i]

        // W(x) <-- W(x)*x^8
        temp = W[15];
        for (int j = 15; j >= 1; j--) W[j] = W[j - 1];
        W[1] ^= R1[temp];
        W[0] = R0[temp];
    }

    // H(x)*P[0]�� ���Ѵ�.
    temp = in[0];
    for (int j = 0; j < 16; j++) {
        state[j] = W[j] ^ HT[temp][j];
    }
}

// (GHASH)====
void GHASH_TableVersion(byte msg[], int msg_blocks,
    byte HT[256][16], byte R0[256], byte R1[256], byte tag[16]) {
    byte x[16];
    byte out[16] = { 0, };
    for (int i = 0; i < msg_blocks; i++) {
        for (int j = 0; j < 16; j++) x[j] = msg[i * 16 + j];
        xor_b_array(out, 16, x);
        //GF128_mul(out, H);
        GF128_Hmul(out, HT, R0, R1);
    }
    for (int j = 0; j < 16; j++) tag[j] = out[j];
}

void Make_GHASH_H_table(byte H[16], byte HT[256][16]) {
    byte Z[16], H_mul[16];
    byte qi_bit;

    for (int i = 0; i < 256; i++) { // 7�� ���� ���׽� 0000 0000 ....  1111 1111
        for (int j = 0; j < 16; j++) {  //��� ������ ���� �ʰ�ȭ
            Z[j] = 0x00;
            H_mul[j] = H[j];
        }
        for (int j = 0; j < 8; j++) { // q0, q1, ... q7
            qi_bit = ((i >> (7 - j)) & 0x01) == 1 ? 0x01 : 0x00;
            if (qi_bit == 1) {
                //for (int k = 0; k < 16; k++) Z[k] ^= H_mul[k];
                xor_b_array(Z, 16, H_mul);
            }
            GF128_xtime(H_mul);
        }
        //for (int k = 0; k < 16; k++) HT[i][k] = Z[k];
        copy_b_array(Z, 16, HT[i]);
    }
}

//(GCM mode)====
void LEA_GCM(byte PT[], int pt_len, byte CTR[16], byte key[16], byte A[], int A_len, byte CT[], byte tag[16]) {  //week13-v2 tag[16] �߰�
    long long int Alen, Clen;
    Alen = (long long int)A_len * 8;
    Clen = (long long int)pt_len * 8;
    uint32 rk[24][6];

    byte first_block[16] = { 0, };
    byte last_block[16];
    if (A_len > 0) { // A_len = 0,1,2,..., 16 (����Ʈ)
        for (int j = 0; j < A_len; j++) first_block[j] = A[j];
    }
    for (int j = 0; j < 8; j++) {
        last_block[j] = (Alen >> (8 * (7 - j))) & 0xff;
        last_block[8 + j] = (Clen >> (8 * (7 - j))) & 0xff;
    }


    byte CTR1[16]; // CTR��� ��ȣȭ�� ���� �����߰�
    byte Y[16]; // �±׸��鶧 XOR�� ���� Y
    byte H[16]; // GHASH�� H
    byte Zero[16] = { 0, }; // ���� ���� ��
    LEA32_Enc_KeySchedule(key, rk);
    LEA32_Encrypt(CTR, rk, Y);
    LEA32_Encrypt(Zero, rk, H);

    copy_b_array(CTR, 16, CTR1);
    counter_inc(CTR1);
    AES_CTR(PT, pt_len, key, CTR1, CT);

    int msg_len, remainder;
    msg_len = (pt_len % 16) == 0 ? pt_len + 2 * 16 : (pt_len / 16) * 16 + 3 * 16;
    remainder = (pt_len % 16) == 0 ? 0 : 16 - (pt_len % 16);

    byte* MSG;
    MSG = (byte*)malloc(msg_len);
    for (int i = 0; i < 16; i++) MSG[i] = first_block[i];
    for (int i = 0; i < pt_len; i++) MSG[16 + i] = CT[i];
    for (int i = 0; i < remainder; i++) MSG[16 + pt_len + i] = 0x00;
    for (int i = 0; i < 16; i++) MSG[16 + pt_len + remainder + i] = last_block[i];

    //(debug) print MSG
    printf("Input for GHASH =");
    for (int i = 0; i < msg_len; i++) {
        printf("%02x", MSG[i]);
        if ((i % 16) == 15) printf(" ");
    }
    printf("\n");

    //GHASH(MSG, msg_len / 16, H, tag);

    byte HT[256][16];
    Make_GHASH_H_table(H, HT);
    GHASH_TableVersion(MSG, msg_len / 16, HT, R0, R1, tag);
    xor_b_array(tag, 16, Y);

}
// '8' --> 8,   'd' --> 13, 'g' --> error!
// �Է� ch : {'0','1',...,'9', 'A', .. ,'F', 'a', ... ,'f'}
// ���: ���ڷ� ��ȯ (0..15) (0..f)
byte Hex2Digit(char ch) {

    if ((ch >= '0') && (ch <= '9')) {
        return  ch - '0'; // ��: '7' - '4' = 3
    }
    else if ((ch >= 'A') && (ch <= 'F')) {
        return ch - 'A' + 10;
    }
    else if ((ch >= 'a') && (ch <= 'f')) {
        return ch - 'a' + 10;
    }
    return -1;
}
// "8d" --> 8d = 8*16 + d = 8*16 + 13 = ???
// h[0] = '8', h[1] = 'd'
byte Hex2Byte(const char h[2]) { // h[0], h[1]
    byte upper, lower; //����(x), ���� ����Ʈ(y) -->  ����Ʈ(xy) 
    upper = h[0];
    lower = h[1];

    return Hex2Digit(upper) * 16 + Hex2Digit(lower);
}

// "8d2e60365f17c7df1040d7501b4a7b5a" --> {8d, 2e, ... , 5a}
// ���ڿ� --> ����Ʈ �迭
// hex_len : ���ڿ��� ���� --> ����Ʈ �迭�� ũ�� = hex_len/2
// byte barr[] : ����Ʈ �迭�� �޸𸮴� �Լ�ȣ�� ���� Ȯ���ؾ� ��
void Hex2Array(const char hex_str[], int hex_len, byte barr[]) {
    //void Hex2Array(const char* hex_str, int hex_len, byte* barr) {
    char h[2];
    byte b_value;
    for (int i = 0; i < hex_len / 2; i++) {
        h[0] = hex_str[2 * i];
        h[1] = hex_str[2 * i + 1];
        b_value = Hex2Byte(h); // {h[0], h[1]} --> h[0]h[1]
        barr[i] = b_value;
    }
}

// ����Ʈ �迭 ����ϱ�
// b_arr[] : ����Ʈ �迭
// len : �迭�� ũ��
// pStr : �߰��� ����� ���ڿ� (default(�⺻��) = nullptr)
void print_b_array(byte b_arr[], int len, const char* pStr) {
    if (pStr != nullptr) {
        printf("%s = ", pStr);
    }
    for (int i = 0; i < len; i++) {
        printf("%02x ", b_arr[i]);
    }
    printf("\n");
}

//=====================
void AES_GCM_testvector0() {  // OK!
    const char* hex_key = "11754cd72aec309bf52f7687212e8957";   //128
    const char* hex_iv = "3c819d9a9bed087615030b65";         //96
    const char* hex_pt = "";   //0
    const char* hex_aad = "";   //0
    const char* hex_ct = "";   //0
    //Tag = 250327c674aaf477aef2675748cf6971

    byte key[16], iv[16], pt[16], ct[16], aad[16], tag[16];

    Hex2Array(hex_key, 32, key);
    Hex2Array(hex_iv, 24, iv);
    //Hex2Array(hex_pt, 32, pt);
    //Hex2Array(hex_aad, 32, aad);

    printf("TestVector-GCM... \n");

    byte CTR0[16] = { 0, };
    for (int i = 0; i < 12; i++) CTR0[i] = iv[i];
    CTR0[15] = 0x01;

    LEA_GCM(pt, 0, CTR0, key, aad, 0, ct, tag);

    //print_b_array(ct, 16, "ct");
    print_b_array(tag, 16, "tag");
}

//=====================
void AES_GCM_testvector1() {
    const char* hex_key = "77be63708971c4e240d1cb79e8d77feb";   //128
    const char* hex_iv = "e0e00f19fed7ba0136a797f3";         //96
    const char* hex_pt = "";   //0
    const char* hex_aad = "7a43ec1d9c0a5a78a0b16533a6213cab";   //128
    const char* hex_ct = "";   //0   
    // Tag = 209fcc8d3675ed938e9c7166709dd946

    byte key[16], iv[16], pt[16], ct[16], aad[16], tag[16];

    Hex2Array(hex_key, 32, key);
    Hex2Array(hex_iv, 24, iv);
    //Hex2Array(hex_pt, 32, pt);
    Hex2Array(hex_aad, 32, aad);

    printf("TestVector-GCM... \n");

    byte CTR0[16] = { 0, };
    for (int i = 0; i < 12; i++) CTR0[i] = iv[i];
    CTR0[15] = 0x01;

    LEA_GCM(pt, 0, CTR0, key, aad, 16, ct, tag);

    //print_b_array(ct, 16, "ct");
    print_b_array(tag, 16, "tag");
}

//=====================
void AES_GCM_testvector2() {
    const char* hex_key = "c939cc13397c1d37de6ae0e1cb7c423c";   //128
    const char* hex_iv = "b3d8cc017cbb89b39e0f67e2";         //96
    const char* hex_pt = "c3b3c41f113a31b73d9a5cd432103069";   //128
    const char* hex_aad = "24825602bd12a984e0092d3e448eda5f";   //128
    const char* hex_ct = "93fe7d9e9bfd10348a5606e5cafa7354";   //128
    // Tag = 0032a1dc85f1c9786925a2e71d8272dd

    byte key[16], iv[16], pt[16], ct[16], aad[16], tag[16];

    Hex2Array(hex_key, 32, key);
    Hex2Array(hex_iv, 24, iv);
    Hex2Array(hex_pt, 32, pt);
    Hex2Array(hex_aad, 32, aad);

    printf("TestVector-GCM... \n");

    byte CTR0[16] = { 0, };
    for (int i = 0; i < 12; i++) CTR0[i] = iv[i];
    CTR0[15] = 0x01;

    LEA_GCM(pt, 16, CTR0, key, aad, 16, ct, tag);

    print_b_array(ct, 16, "ct");
    print_b_array(tag, 16, "tag");
}

//=====================
void AES_CTR_testvector() { //OK-CTR
    const char* hex_key = "2b7e151628aed2a6abf7158809cf4f3c";
    const char* hex_iv = "f0f1f2f3f4f5f6f7f8f9fafbfcfdfeff";
    const char* hex_pt = "6bc1bee22e409f96e93d7e117393172a";
    const char* hex_ct = "874d6191b620e3261bef6864990db6ce";

    byte key[16], iv[16], pt[16], ct[16];

    Hex2Array(hex_key, 32, key);
    Hex2Array(hex_iv, 32, iv);
    Hex2Array(hex_pt, 32, pt);

    printf("TestVector-CTR... \n");

    AES_CTR(pt, 16, key, iv, ct);

    print_b_array(ct, 16, "ct");
}

void AES_GCM_testvector3() {
    /*
    [Keylen = 128]
    [IVlen = 96]
    [PTlen = 128]
    [AADlen = 0]
    [Taglen = 128]

    Key = 7fddb57453c241d03efbed3ac44e371c
    IV = ee283a3fc75575e33efd4887
    PT = d5de42b461646c255c87bd2962d3b9a2
    AAD =
    CT = 2ccda4a5415cb91e135c2a0f78c9b2fd
    Tag = b36d1df9b9d5e596f83e8b7f52971cb3
    */
    const char* hex_key = "7fddb57453c241d03efbed3ac44e371c";
    const char* hex_iv = "ee283a3fc75575e33efd4887";
    const char* hex_pt = "d5de42b461646c255c87bd2962d3b9a2";
    const char* hex_aad = "";
    const char* hex_ct = "2ccda4a5415cb91e135c2a0f78c9b2fd";
    const char* hex_tag = "b36d1df9b9d5e596f83e8b7f52971cb3";


    byte key[16], iv[16], pt[16], ct[16], aad[16], tag[16];

    Hex2Array(hex_key, 32, key);
    Hex2Array(hex_iv, 24, iv);  //96��Ʈ
    Hex2Array(hex_pt, 32, pt);
    //Hex2Array(hex_aad, 32, aad);

    printf("TestVector-GCM... \n");

    // CTR = IV(96bits) || 00...01 (32bits)
    byte CTR[16] = { 0, };
    for (int i = 0; i < 12; i++) CTR[i] = iv[i];
    CTR[15] = 0x01;

    LEA_GCM(pt, 16, CTR, key, aad, 0, ct, tag);

    print_b_array(ct, 16, "(calculated) ct = ");
    printf("(expected) ct = %s\n", hex_ct);
    print_b_array(tag, 16, "(calculated) tag = ");
    printf("(expected) tag = %s\n", hex_tag);
}

int main()
{
    AES_GCM_testvector0();
    AES_GCM_testvector1();
    AES_GCM_testvector2();
    AES_GCM_testvector3();
}